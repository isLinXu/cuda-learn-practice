//
// Created by gatilin on 2023/11/16.
//

#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int N = 1 << 20;
    size_t bytes = N * sizeof(int);

    int *a, *b, *c;
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    vectorAdd<<<gridSize, blockSize>>>(a, b, c, N);
    hipDeviceSynchronize();

    // Verify the result
    for (int i = 0; i < N; i++) {
        assert(c[i] == a[i] + b[i]);
    }

    std::cout << "COMPLETED SUCCESSFULLY\n";

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}